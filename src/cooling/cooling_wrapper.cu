
#include <hip/hip_runtime.h>
/*! \file cooling_wrapper.cu
 *  \brief Wrapper file for to load CUDA cooling tables. */

#ifdef CUDA
#ifdef CLOUDY_COOL

#include <stdio.h>
#include <stdlib.h>
#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../cooling/cooling_wrapper.h"
#include "../cooling/cooling_cuda.h"

cudaTextureObject_t coolTexObj = 0;
cudaTextureObject_t heatTexObj = 0;
//texture<float, 2, cudaReadModeElementType> coolTexObj;
//texture<float, 2, cudaReadModeElementType> heatTexObj;
cudaArray* cuCoolArray;
cudaArray* cuHeatArray;


/* \fn void Host_Read_Cooling_Tables(float* cooling_table, float* heating_table)
 * \brief Load the Cloudy cooling tables into host (CPU) memory. */
void Host_Read_Cooling_Tables(float* cooling_table, float* heating_table)
{
  double *n_arr;
  double *T_arr;
  double *L_arr;
  double *H_arr;

  int i;
  int nx = 121;
  int ny = 81;

  FILE *infile;
  char buffer[0x1000];
  char * pch;

  // allocate arrays for temperature data
  n_arr = (double *) malloc(nx*ny*sizeof(double));
  T_arr = (double *) malloc(nx*ny*sizeof(double));
  L_arr = (double *) malloc(nx*ny*sizeof(double));
  H_arr = (double *) malloc(nx*ny*sizeof(double));

  // Read in cloudy cooling/heating curve (function of density and temperature)
  i=0;
  const char* cloudy_filename = "src/cooling/cloudy_coolingcurve.txt";
  infile = fopen(cloudy_filename, "r");
  if (infile == NULL) {
    printf("Unable to open Cloudy file with expected path: %s\n", cloudy_filename);
    exit(1);
  }
  while (fgets(buffer, sizeof(buffer), infile) != NULL)
  {
    if (buffer[0] == '#') {
      continue;
    }
    else {
      pch = strtok(buffer, "\t");
      n_arr[i] = atof(pch);
      while (pch != NULL)
      {
        pch = strtok(NULL, "\t");
        if (pch != NULL)
          T_arr[i] = atof(pch);
        pch = strtok(NULL, "\t");
        if (pch != NULL)
          L_arr[i] = atof(pch);
        pch = strtok(NULL, "\t");
        if (pch != NULL)
          H_arr[i] = atof(pch);
      }
      i++;
    }
  }
  fclose(infile);

  // copy data from cooling array into the table
  for (i=0; i<nx*ny; i++)
  {
    cooling_table[i] = float(L_arr[i]);
    heating_table[i] = float(H_arr[i]);
  }

  // Free arrays used to read in table data
  free(n_arr);
  free(T_arr);
  free(L_arr);
  free(H_arr);
}


/* \fn void Load_Cuda_Textures2()
 * \brief Load the Cloudy cooling tables into texture memory on the GPU. */
void Load_Cuda_Textures()
{
  printf("Initializing Cloudy Textures");
  float *cooling_table;
  float *heating_table;
  const int nx = 81;
  const int ny = 121;

  // allocate host arrays to be copied to textures
  // these arrays are declared as external pointers in global.h
  CudaSafeCall( cudaHostAlloc(&cooling_table, nx*ny*sizeof(float), cudaHostAllocDefault) );
  CudaSafeCall( cudaHostAlloc(&heating_table, nx*ny*sizeof(float), cudaHostAllocDefault) );

  // Read cooling tables into the host arrays
  Host_Read_Cooling_Tables(cooling_table, heating_table);

  // Allocate CUDA arrays in device memory
  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatKindFloat);
  cudaMallocArray(&cuCoolArray, &channelDesc, nx, ny);
  cudaMallocArray(&cuHeatArray, &channelDesc, nx, ny);

  // Copy the cooling and heating arrays from host to device
  // in host memory
  cudaMemcpyToArray(cuCoolArray, 0, 0, cooling_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);
  cudaMemcpyToArray(cuHeatArray, 0, 0, heating_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);
  //cudaMemcpy(cuCoolArray, cooling_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);
  //cudaMemcpy(cuHeatArray, heating_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);

  // Specify textures
  struct cudaResourceDesc coolResDesc;
  memset(&coolResDesc, 0, sizeof(coolResDesc));
  coolResDesc.resType = cudaResourceTypeArray;
  coolResDesc.res.array.array = cuCoolArray;
  struct cudaResourceDesc heatResDesc;
  memset(&heatResDesc, 0, sizeof(heatResDesc));
  heatResDesc.resType = cudaResourceTypeArray;
  heatResDesc.res.array.array = cuHeatArray;  

  // Specify texture object parameters (same for both tables)
  struct cudaTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  texDesc.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  texDesc.filterMode = cudaFilterModeLinear;
  texDesc.readMode = cudaReadModeElementType;
  texDesc.normalizedCoords = 1;

  // Create texture objects
  cudaCreateTextureObject(&coolTexObj, &coolResDesc, &texDesc, NULL);
  cudaCreateTextureObject(&heatTexObj, &heatResDesc, &texDesc, NULL);

  // Free the memory associated with the cooling tables on the host
  CudaSafeCall( cudaFreeHost(cooling_table) );
  CudaSafeCall( cudaFreeHost(heating_table) );

}


/* \fn void Load_Cuda_Textures()
 * \brief Load the Cloudy cooling tables into texture memory on the GPU. */
/*
void Load_Cuda_Textures()
{

  float *cooling_table;
  float *heating_table;
  const int nx = 81;
  const int ny = 121;

  // allocate host arrays to be copied to textures
  // these arrays are declared as external pointers in global.h
  CudaSafeCall( cudaHostAlloc(&cooling_table, nx*ny*sizeof(float), cudaHostAllocDefault) );
  CudaSafeCall( cudaHostAlloc(&heating_table, nx*ny*sizeof(float), cudaHostAllocDefault) );

  // Load cooling tables into the host arrays
  Load_Cooling_Tables(cooling_table, heating_table);

  // Allocate CUDA arrays in device memory
  cudaChannelFormatDesc channelDesc = cudaCreateChannelDesc(32, 0, 0, 0, cudaChannelFormatKindFloat);
  cudaMallocArray(&cuCoolArray, &channelDesc, nx, ny);
  cudaMallocArray(&cuHeatArray, &channelDesc, nx, ny);
  // Copy to device memory the cooling and heating arrays
  // in host memory
  cudaMemcpyToArray(cuCoolArray, 0, 0, cooling_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);
  cudaMemcpyToArray(cuHeatArray, 0, 0, heating_table, nx*ny*sizeof(float), cudaMemcpyHostToDevice);


  // Specify texture reference parameters (same for both tables)
  coolTexObj.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  coolTexObj.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  coolTexObj.filterMode = cudaFilterModeLinear; // bi-linear interpolation
  coolTexObj.normalized = true;
  heatTexObj.addressMode[0] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  heatTexObj.addressMode[1] = cudaAddressModeClamp; // out-of-bounds fetches return border values
  heatTexObj.filterMode = cudaFilterModeLinear; // bi-linear interpolation
  heatTexObj.normalized = true;

  cudaBindTextureToArray(coolTexObj, cuCoolArray);
  cudaBindTextureToArray(heatTexObj, cuHeatArray);

  // Free the memory associated with the cooling tables on the host
  CudaSafeCall( cudaFreeHost(cooling_table) );
  CudaSafeCall( cudaFreeHost(heating_table) );

}
*/




void Free_Cuda_Textures()
{
  // unbind the cuda textures
  // cudaUnbindTexture(coolTexObj);
  // cudaUnbindTexture(heatTexObj);
  cudaDestroyTextureObject(coolTexObj);
  cudaDestroyTextureObject(heatTexObj);

  // Free the device memory associated with the cuda arrays
  cudaFreeArray(cuCoolArray);
  cudaFreeArray(cuHeatArray);

}


#endif
#endif
