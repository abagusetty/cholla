#include "hip/hip_runtime.h"
/*!
* \file cooling_cuda_tests.cpp
* \author Helena Richie (helenarichie@pitt.edu)
* \brief Testing the CIE cooling rate function in cooling_cuda.cu
*
*/

// STL Includes
#include <iostream>
#include <string>
#include <math.h>

// External Includes
#include <gtest/gtest.h> // Include GoogleTest and related libraries/headers

// Local Includes
#include "../global/global_cuda.h"
#include "../utils/gpu.hpp"
#include "../utils/testing_utilities.h"
#include "../cooling/cooling_cuda.h" // Include code to test

#ifdef COOLING_GPU

TEST(tCOOLINGTestCIECool, TestCIECoolExpectCorrectOutput) // test suite name, test name
{
    Real const testn = 1;
    Real const testT = pow(10, 5.0);
    Real const testNumber = CIE_cool(testn, testT);

    Real const fiducialNumber = 4.6639082688443984*pow(10, -22);

    double absoluteDiff;
    int64_t ulpsDiff;

    bool isTrue;

    isTrue = testingUtilities::nearlyEqualDbl(fiducialNumber, testNumber, absoluteDiff, ulpsDiff);
    
    EXPECT_TRUE(isTrue)
            << "The fiducial value is:       " << fiducialNumber  << std::endl
            << "The test value is:           " << testNumber      << std::endl
            << "The absolute difference is:  " << absoluteDiff    << std::endl
            << "The ULP difference is:       " << ulpsDiff        << std::endl;
}

#endif // COOLING