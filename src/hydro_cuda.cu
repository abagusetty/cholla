#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
/*! \file hydro_cuda.cu
 *  \brief Definitions of functions used in all cuda integration algorithms. */
#ifdef CUDA

#include<stdio.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include"global.h"
#include"global_cuda.h"
#include"hydro_cuda.h"
#include"gravity_cuda.h"


__global__ void Update_Conserved_Variables_1D(Real *dev_conserved, Real *dev_F, int n_cells, int x_off, int n_ghost, Real dx, Real xbound, Real dt, Real gamma, int n_fields)
{
  int id;
  #ifdef STATIC_GRAV
  Real d, d_inv, vx;  
  Real gx, d_n, d_inv_n, vx_n;
  gx = 0.0;
  #endif
  
  Real dtodx = dt/dx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    #ifdef STATIC_GRAV
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    #endif
  
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F[            id-1] - dev_F[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F[  n_cells + id-1] - dev_F[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F[2*n_cells + id-1] - dev_F[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F[3*n_cells + id-1] - dev_F[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F[4*n_cells + id-1] - dev_F[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F[(5+i)*n_cells + id-1] - dev_F[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F[(n_fields-1)*n_cells + id-1] - dev_F[(n_fields-1)*n_cells + id]);
    #endif
    #ifdef STATIC_GRAV // add gravitational source terms, time averaged from n to n+1
    calc_g_1D(id, x_off, n_ghost, dx, xbound, &gx);    
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n);
    #endif    
    if (dev_conserved[id] != dev_conserved[id]) printf("%3d Thread crashed in final update. %f\n", id, dev_conserved[id]);
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%d Negative pressure after final update.\n", id);
    */
  }


}


__global__ void Update_Conserved_Variables_2D(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y, int nx, int ny, int x_off, int y_off, int n_ghost, Real dx, Real dy, Real xbound, Real ybound, Real dt, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  int imo, jmo;

  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy;
  Real gx, gy, d_n, d_inv_n, vx_n, vy_n;
  gx = 0.0;
  gy = 0.0;
  #endif

  Real dtodx = dt/dx;
  Real dtody = dt/dy;

  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  imo = xid-1 + yid*nx;
  jmo = xid + (yid-1)*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    #ifdef STATIC_GRAV
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    #endif
    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id]) 
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id]) 
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id]); 
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                        +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id]);
    }
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id]);
    #endif
    #ifdef STATIC_GRAV 
    // calculate the gravitational acceleration as a function of x & y position
    calc_g_2D(xid, yid, x_off, y_off, n_ghost, dx, dy, xbound, ybound, &gx, &gy);
    // add gravitational source terms, time averaged from n to n+1                                 
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n);
    #endif
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id]) {
      printf("%3d %3d Thread crashed in final update. %f %f %f\n", xid, yid, dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dev_conserved[id]);
    }   
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0)
      printf("%3d %3d Negative pressure after final update. %f %f %f %f\n", xid, yid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, P);    
    */
  }

}



__global__ void Update_Conserved_Variables_3D(Real *dev_conserved, Real *dev_F_x, Real *dev_F_y,  Real *dev_F_z,
                                              int nx, int ny, int nz, int x_off, int y_off, int z_off, int n_ghost, 
                                              Real dx, Real dy, Real dz, Real xbound, Real ybound, Real zbound, Real dt,
                                              Real gamma, int n_fields)
{
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;
  #ifdef STATIC_GRAV
  Real d, d_inv, vx, vy, vz;
  Real gx, gy, gz, d_n, d_inv_n, vx_n, vy_n, vz_n;
  gx = 0.0;
  gy = 0.0;
  gz = 0.0;
  #endif

  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  imo = xid-1 + yid*nx + zid*nx*ny;
  jmo = xid + (yid-1)*nx + zid*nx*ny;
  kmo = xid + yid*nx + (zid-1)*nx*ny;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    #ifdef STATIC_GRAV
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    #endif

    // update the conserved variable array
    dev_conserved[            id] += dtodx * (dev_F_x[            imo] - dev_F_x[            id])
                                  +  dtody * (dev_F_y[            jmo] - dev_F_y[            id])
                                  +  dtodz * (dev_F_z[            kmo] - dev_F_z[            id]);
    dev_conserved[  n_cells + id] += dtodx * (dev_F_x[  n_cells + imo] - dev_F_x[  n_cells + id])
                                  +  dtody * (dev_F_y[  n_cells + jmo] - dev_F_y[  n_cells + id])
                                  +  dtodz * (dev_F_z[  n_cells + kmo] - dev_F_z[  n_cells + id]);
    dev_conserved[2*n_cells + id] += dtodx * (dev_F_x[2*n_cells + imo] - dev_F_x[2*n_cells + id])
                                  +  dtody * (dev_F_y[2*n_cells + jmo] - dev_F_y[2*n_cells + id])
                                  +  dtodz * (dev_F_z[2*n_cells + kmo] - dev_F_z[2*n_cells + id]);
    dev_conserved[3*n_cells + id] += dtodx * (dev_F_x[3*n_cells + imo] - dev_F_x[3*n_cells + id])
                                  +  dtody * (dev_F_y[3*n_cells + jmo] - dev_F_y[3*n_cells + id])
                                  +  dtodz * (dev_F_z[3*n_cells + kmo] - dev_F_z[3*n_cells + id]);
    dev_conserved[4*n_cells + id] += dtodx * (dev_F_x[4*n_cells + imo] - dev_F_x[4*n_cells + id])
                                  +  dtody * (dev_F_y[4*n_cells + jmo] - dev_F_y[4*n_cells + id])
                                  +  dtodz * (dev_F_z[4*n_cells + kmo] - dev_F_z[4*n_cells + id]);
    #ifdef SCALAR
    for (int i=0; i<NSCALARS; i++) {
      dev_conserved[(5+i)*n_cells + id] += dtodx * (dev_F_x[(5+i)*n_cells + imo] - dev_F_x[(5+i)*n_cells + id])
                                    +  dtody * (dev_F_y[(5+i)*n_cells + jmo] - dev_F_y[(5+i)*n_cells + id])
                                    +  dtodz * (dev_F_z[(5+i)*n_cells + kmo] - dev_F_z[(5+i)*n_cells + id]);
    }                              
    #endif
    #ifdef DE
    dev_conserved[(n_fields-1)*n_cells + id] += dtodx * (dev_F_x[(n_fields-1)*n_cells + imo] - dev_F_x[(n_fields-1)*n_cells + id])
                                  +  dtody * (dev_F_y[(n_fields-1)*n_cells + jmo] - dev_F_y[(n_fields-1)*n_cells + id])
                                  +  dtodz * (dev_F_z[(n_fields-1)*n_cells + kmo] - dev_F_z[(n_fields-1)*n_cells + id]);
                                  // +  0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
                                  //Note: this term is added in a separate kernel to avoid syncronization issues
    #endif
    #ifdef STATIC_GRAV 
    calc_g_3D(xid, yid, zid, x_off, y_off, z_off, n_ghost, dx, dy, dz, xbound, ybound, zbound, &gx, &gy, &gz);
    d_n  =  dev_conserved[            id];
    d_inv_n = 1.0 / d_n;
    vx_n =  dev_conserved[1*n_cells + id] * d_inv_n;
    vy_n =  dev_conserved[2*n_cells + id] * d_inv_n;
    vz_n =  dev_conserved[3*n_cells + id] * d_inv_n;
    dev_conserved[  n_cells + id] += 0.5*dt*gx*(d + d_n);
    dev_conserved[2*n_cells + id] += 0.5*dt*gy*(d + d_n);
    dev_conserved[3*n_cells + id] += 0.5*dt*gz*(d + d_n);
    dev_conserved[4*n_cells + id] += 0.25*dt*gx*(d + d_n)*(vx + vx_n)
                                  +  0.25*dt*gy*(d + d_n)*(vy + vy_n)
                                  +  0.25*dt*gz*(d + d_n)*(vz + vz_n);
    #endif    
    if (dev_conserved[id] < 0.0 || dev_conserved[id] != dev_conserved[id] || dev_conserved[4*n_cells + id] < 0.0 || dev_conserved[4*n_cells+id] != dev_conserved[4*n_cells+id]) {
      printf("%3d %3d %3d Thread crashed in final update. %e %e %e %e %e\n", xid+x_off, yid+y_off, zid+z_off, dev_conserved[id], dtodx*(dev_F_x[imo]-dev_F_x[id]), dtody*(dev_F_y[jmo]-dev_F_y[id]), dtodz*(dev_F_z[kmo]-dev_F_z[id]), dev_conserved[4*n_cells+id]);
    }
    /*
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    if (P < 0.0) printf("%3d %3d %3d Negative pressure after final update. %f %f %f %f %f\n", xid, yid, zid, dev_conserved[4*n_cells + id], 0.5*d*vx*vx, 0.5*d*vy*vy, 0.5*d*vz*vz, P);
    */
  }

}





__global__ void Calc_dt_1D(Real *dev_conserved, int n_cells, int n_ghost, Real dx, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();


  // threads corresponding to real cells do the calculation
  if (id > n_ghost - 1 && id < n_cells-n_ghost)
  {
    // start timestep calculation here
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) TINY_NUMBER);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = (fabs(vx)+cs)/dx;
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];


}



__global__ void Calc_dt_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real dx, Real dy, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, P, cs;
  int id, tid, xid, yid, n_cells;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  // and a thread id within the block
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    P  = (dev_conserved[4*n_cells + id] - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    P  = fmax(P, (Real) 1.0e-20);
    // find the max wavespeed in that cell, use it to calculate the inverse timestep
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockId] = max_dti[0];

}


__global__ void Calc_dt_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real dx, Real dy, Real dz, Real *dti_array, Real gamma)
{
  __shared__ Real max_dti[TPB];

  Real d, d_inv, vx, vy, vz, E, P, cs;
  int id, xid, yid, zid, n_cells;
  int tid;

  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  // and a thread id within the block  
  tid = threadIdx.x;

  // set shared memory to 0
  max_dti[tid] = 0;
  __syncthreads();

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  = dev_conserved[4*n_cells + id];
    P  = (E - 0.5*d*(vx*vx + vy*vy + vz*vz)) * (gamma - 1.0);
    cs = sqrt(d_inv * gamma * P);
    max_dti[tid] = fmax((fabs(vx)+cs)/dx, (fabs(vy)+cs)/dy);
    max_dti[tid] = fmax(max_dti[tid], (fabs(vz)+cs)/dz);
    max_dti[tid] = fmax(max_dti[tid], 0.0);
  }
  __syncthreads();
  
  // do the reduction in shared memory (find the max inverse timestep in the block)
  for (unsigned int s=1; s<blockDim.x; s*=2) {
    if (tid % (2*s) == 0) {
      max_dti[tid] = fmax(max_dti[tid], max_dti[tid + s]);
    }
    __syncthreads();
  }

  // write the result for this block to global memory
  if (tid == 0) dti_array[blockIdx.x] = max_dti[0];

}

#ifdef DE
__host__ __device__ Real Get_Pressure_From_DE( Real E, Real U_total, Real U_advected, Real gamma ){
  
  Real U, P;
  Real eta = DE_ETA_1;
  
  // Apply same condition as Byan+2013 to select the internal energy from which compute pressure.
  if( U_total / E > eta ) U = U_total;
  else U = U_advected;
  
  P = U * (gamma - 1.0);
  return P;
}

__global__ void Partial_Update_Advected_Internal_Energy_1D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, int nx, int n_ghost, Real dx, Real dt, Real gamma, int n_fields ){
  
  int id, xid, n_cells;
  int imo, ipo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo;
  Real  P, E, E_kin, GE;
  
  
  Real dtodx = dt/dx;
  n_cells = nx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;

  
  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = Get_Pressure_From_DE( E, E - E_kin, GE, gamma );  
    P  = fmax(P, (Real) TINY_NUMBER); 

    imo = xid-1;
    ipo = xid+1;
    
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    
    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo));
 
  }  
}


__global__ void Partial_Update_Advected_Internal_Energy_2D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, int nx, int ny, int n_ghost, Real dx, Real dy, Real dt, Real gamma, int n_fields ){
  
  int id, xid, yid, n_cells;
  int imo, jmo;
  int ipo, jpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo;
  Real  P, E, E_kin, GE;
  
  
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  
  
  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = Get_Pressure_From_DE( E, E - E_kin, GE, gamma );  
    P  = fmax(P, (Real) TINY_NUMBER); 

    imo = xid-1 + yid*nx;
    ipo = xid+1 + yid*nx;
    jmo = xid + (yid-1)*nx;
    jpo = xid + (yid+1)*nx;
    
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    
    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo));
 
  }  
}

__global__ void Partial_Update_Advected_Internal_Energy_3D( Real *dev_conserved, Real *Q_Lx, Real *Q_Rx, Real *Q_Ly, Real *Q_Ry, Real *Q_Lz, Real *Q_Rz, int nx, int ny, int nz,  int n_ghost, Real dx, Real dy, Real dz,  Real dt, Real gamma, int n_fields ){
  
  int id, xid, yid, zid, n_cells;
  int imo, jmo, kmo;
  int ipo, jpo, kpo;
  Real d, d_inv, vx, vy, vz;
  Real vx_imo, vx_ipo, vy_jmo, vy_jpo, vz_kmo, vz_kpo;
  Real  P, E, E_kin, GE;
  // Real vx_L, vx_R, vy_L, vy_R, vz_L, vz_R;
  
  
  Real dtodx = dt/dx;
  Real dtody = dt/dy;
  Real dtodz = dt/dz;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  
  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    //PRESSURE_DE
    E = dev_conserved[4*n_cells + id];
    GE = dev_conserved[(n_fields-1)*n_cells + id];
    E_kin = 0.5 * d * ( vx*vx + vy*vy + vz*vz );
    P = Get_Pressure_From_DE( E, E - E_kin, GE, gamma );  
    P  = fmax(P, (Real) TINY_NUMBER); 

    imo = xid-1 + yid*nx + zid*nx*ny;
    jmo = xid + (yid-1)*nx + zid*nx*ny;
    kmo = xid + yid*nx + (zid-1)*nx*ny;
    
    ipo = xid+1 + yid*nx + zid*nx*ny;
    jpo = xid + (yid+1)*nx + zid*nx*ny;
    kpo = xid + yid*nx + (zid+1)*nx*ny;
    
    vx_imo = dev_conserved[1*n_cells + imo] / dev_conserved[imo]; 
    vx_ipo = dev_conserved[1*n_cells + ipo] / dev_conserved[ipo]; 
    vy_jmo = dev_conserved[2*n_cells + jmo] / dev_conserved[jmo]; 
    vy_jpo = dev_conserved[2*n_cells + jpo] / dev_conserved[jpo]; 
    vz_kmo = dev_conserved[3*n_cells + kmo] / dev_conserved[kmo]; 
    vz_kpo = dev_conserved[3*n_cells + kpo] / dev_conserved[kpo];
    
    // Use center values of neighbor cells for the divergence of velocity
    dev_conserved[(n_fields-1)*n_cells + id] += 0.5*P*(dtodx*(vx_imo-vx_ipo) + dtody*(vy_jmo-vy_jpo) + dtodz*(vz_kmo-vz_kpo));
 
    // OPTION 2: Use the reconstrcted velocities to compute the velocity gradient
    //Use the reconstructed Velocities instead of neighbor cells centered values 
    // vx_R = Q_Lx[1*n_cells + id]  / Q_Lx[id]; 
    // vx_L = Q_Rx[1*n_cells + imo] / Q_Rx[imo]; 
    // vy_R = Q_Ly[2*n_cells + id]  / Q_Ly[id]; 
    // vy_L = Q_Ry[2*n_cells + jmo] / Q_Ry[jmo];
    // vz_R = Q_Lz[3*n_cells + id]  / Q_Lz[id]; 
    // vz_L = Q_Rz[3*n_cells + kmo] / Q_Rz[kmo]; 
    
    //Use the reconstructed Velocities instead of neighbor cells centered values
    // dev_conserved[(n_fields-1)*n_cells + id] +=  P * ( dtodx * ( vx_L - vx_R ) + dtody * ( vy_L - vy_R ) + dtodz * ( vz_L - vz_R ) );

    
  }  
}


__global__ void Select_Internal_Energy_1D( Real *dev_conserved, int nx, int n_ghost, int n_fields ){
  
  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo;
  n_cells = nx;
  
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;
  
  imo = max(xid-1, n_ghost);
  imo = imo;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );
    
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
  
    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;
    
    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);
    
    //Write Selected internal energy to the GasEnergy array ONLY 
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;
  
  }
}


__global__ void Select_Internal_Energy_2D( Real *dev_conserved, int nx, int ny, int n_ghost, int n_fields ){
  
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo;
  n_cells = nx*ny;
  
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  
  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );
    
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);
  
    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;
    
    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);
    
    //Write Selected internal energy to the GasEnergy array ONLY 
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;
  
  }
}


__global__ void Select_Internal_Energy_3D( Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields ){
  
  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, E, U_total, U_advected, U, Emax;
  int imo, ipo, jmo, jpo, kmo, kpo;
  n_cells = nx*ny*nz;
  
  Real eta_2 = DE_ETA_2;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;
  
  imo = max(xid-1, n_ghost);
  imo = imo + yid*nx + zid*nx*ny;
  ipo = min(xid+1, nx-n_ghost-1);
  ipo = ipo + yid*nx + zid*nx*ny;
  jmo = max(yid-1, n_ghost);
  jmo = xid + jmo*nx + zid*nx*ny;
  jpo = min(yid+1, ny-n_ghost-1);
  jpo = xid + jpo*nx + zid*nx*ny;
  kmo = max(zid-1, n_ghost);
  kmo = xid + yid*nx + kmo*nx*ny;
  kpo = min(zid+1, nz-n_ghost-1);
  kpo = xid + yid*nx + kpo*nx*ny;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    E  =  dev_conserved[4*n_cells + id];
    U_advected = dev_conserved[(n_fields-1)*n_cells + id];
    U_total = E - 0.5*d*( vx*vx + vy*vy + vz*vz );
    
    //find the max nearby total energy 
    Emax = fmax(dev_conserved[4*n_cells + imo], E);
    Emax = fmax(Emax, dev_conserved[4*n_cells + ipo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + jpo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kmo]);
    Emax = fmax(Emax, dev_conserved[4*n_cells + kpo]);
  
    if (U_total/Emax > eta_2 ) U = U_total;
    else U = U_advected;
    
    //Optional: Avoid Negative Internal  Energies
    U = fmax(U, (Real) TINY_NUMBER);
    
    //Write Selected internal energy to the GasEnergy array ONLY 
    //to avoid mixing updated and non-updated values of E
    //since the Dual Energy condition depends on the neighbour cells
    dev_conserved[(n_fields-1)*n_cells + id] = U;
  
  }
}

__global__ void Sync_Energies_1D(Real *dev_conserved, int nx, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  xid = id;


  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previusly selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }

}


__global__ void Sync_Energies_2D(Real *dev_conserved, int nx, int ny, int n_ghost, Real gamma, int n_fields)
{
  int id, xid, yid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx*ny;

  // get a global thread ID
  int blockId = blockIdx.x + blockIdx.y*gridDim.x;
  id = threadIdx.x + blockId * blockDim.x;
  yid = id / nx;
  xid = id - yid*nx;
  

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previusly selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }

}


__global__ void Sync_Energies_3D(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, Real gamma, int n_fields)
{
  //Called in a separate kernel to avoid interfering with energy selection in Select_Internal_Energy

  int id, xid, yid, zid, n_cells;
  Real d, d_inv, vx, vy, vz, U;
  n_cells = nx*ny*nz;

  // get a global thread ID
  id = threadIdx.x + blockIdx.x * blockDim.x;
  zid = id / (nx*ny);
  yid = (id - zid*nx*ny) / nx;
  xid = id - zid*nx*ny - yid*nx;

  // threads corresponding to real cells do the calculation
  if (xid > n_ghost-1 && xid < nx-n_ghost && yid > n_ghost-1 && yid < ny-n_ghost && zid > n_ghost-1 && zid < nz-n_ghost)
  {
    // every thread collects the conserved variables it needs from global memory
    d  =  dev_conserved[            id];
    d_inv = 1.0 / d;
    vx =  dev_conserved[1*n_cells + id] * d_inv;
    vy =  dev_conserved[2*n_cells + id] * d_inv;
    vz =  dev_conserved[3*n_cells + id] * d_inv;
    U = dev_conserved[(n_fields-1)*n_cells + id];

    //Use the previusly selected Internal Energy to update the total energy
    dev_conserved[4*n_cells + id] = 0.5*d*( vx*vx + vy*vy + vz*vz ) + U;
  }
}


#endif //DE


#endif //CUDA
