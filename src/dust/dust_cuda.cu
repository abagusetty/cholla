
#include <hip/hip_runtime.h>
#ifdef CUDA
#ifdef SCALAR

#include "dust_model.h"

#include <cstdio>
#include<stdio.h>
#include <fstream>

#include <vector>

#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"
#include "../utils/cuda_utilities.h"
#include "../grid/grid3D.h"

int main() {
  Conserved_Init(host_conserved, rho, vx, vy, vz, P, rho_d, gamma, k_n_cells, k_nx, k_ny, k_nz, k_n_ghost, k_n_fields);
}
  

void Dust_Update(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt, Real gamma) {
    dim3 dim1dGrid(k_ngrid, 1, 1);
    dim3 dim1dBlock(TPB, 1, 1);
    hipLaunchKernelGGL(Dust_Kernel, dim1dGrid, dim1dBlock, 0, 0, dev_conserved, nx, ny, nz, n_ghost, n_fields, dt, gamma, params_dev);
    CudaCheckError();  
}

__global__ void Dust_Kernel(Real *dev_conserved, int nx, int ny, int nz, int n_ghost, int n_fields, Real dt, Real gamma) {
    //__shared__ Real min_dt[TPB];
    // get grid indices
    Real const K = 1e30;
    int n_cells = nx * ny * nz;
    int is, ie, js, je, ks, ke;
    cuda_utilities::Get_Real_Indices(n_ghost, nx, ny, nz, is, ie, js, je, ks, ke);
    // get a global thread ID
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int id = threadIdx.x + blockId * blockDim.x;
    int zid = id / (nx * ny);
    int yid = (id - zid * nx * ny) / nx;
    int xid = id - zid * nx * ny - yid * nx;
    // add a thread id within the block 

    // define physics variables
    Real d_gas, d_dust; // fluid mass densities
    Real n = 1; // gas number density
    Real T, E, P; // temperature, energy, pressure
    Real vx, vy, vz; // velocities
    #ifdef DE
    Real ge;
    #endif // DE

    dt *= 3.154e7; // in seconds

    // define integration variables
    Real dd_dt; // instantaneous rate of change in dust density
    Real dd; // change in dust density at current time-step
    Real dd_max = 0.01; // allowable percentage of dust density increase
    Real dt_sub; //refined timestep

    if (xid >= is && xid < ie && yid >= js && yid < je && zid >= ks && zid < ke) {
        // get quantities from dev_conserved
        d_gas = dev_conserved[id];
        //d_dust = dev_conserved[5*n_cells + id];
        d_dust = dev_conserved[5*n_cells + id];
        E = dev_conserved[4*n_cells + id];
        //printf("kernel: %7.4e\n", d_dust);
        // make sure thread hasn't crashed

        // multiply small values by arbitrary constant to preserve precision
        d_gas *= K;
        d_dust *= K;

        if (E < 0.0 || E != E) return;
        
        vx = dev_conserved[1*n_cells + id] / d_gas;
        vy = dev_conserved[2*n_cells + id] / d_gas;
        vz = dev_conserved[3*n_cells + id] / d_gas;

        #ifdef DE
        ge = dev_conserved[(n_fields-1)*n_cells + id] / d_gas;
        ge = fmax(ge, (Real) TINY_NUMBER);
        #endif // DE

        // calculate physical quantities
        P = hydro_utilities::Calc_Pressure_Primitive(E, d_gas, vx, vy, vz, gamma);

        Real T_init;
        T_init = hydro_utilities::Calc_Temp(P, n);

        #ifdef DE
        T_init = hydro_utilities::Calc_Temp_DE(d_gas, ge, gamma, n);
        #endif // DE

        T = T_init;

        Real tau_sp = calc_tau_sp(n, T);

        dd_dt = calc_dd_dt(d_dust, tau_sp);
        dd = dd_dt * dt;

        params_dev[0] = T;
        params_dev[1] = n;
        params_dev[2] = tau_sp/3.154e7;
        params_dev[3] = dd_dt;
        params_dev[4] = dd; 

        // ensure that dust density is not changing too rapidly
        bool time_refine = false;
        while (dd/d_dust > dd_max) {
            time_refine = true;
            dt_sub = dd_max * d_dust / dd_dt;
            d_dust += dt_sub * dd_dt;
            dt -= dt_sub;
            dd_dt = calc_dd_dt(d_dust, tau_sp);
            dd = dt * dd_dt;
        }

        params_dev[5] = time_refine;

        // update dust density
        d_dust += dd;

        // remove scaling constant
        d_gas /= K;
        d_dust /= K;
        dev_conserved[5*n_cells + id] = d_dust;
        
        #ifdef DE
        dev_conserved[(n_fields-1)*n_cells + id] = d*ge;
        #endif
    }
}

__device__ Real calc_tau_sp(Real n, Real T) {
  Real YR_IN_S = 3.154e7;
  Real a1 = 1; // dust grain size in units of 0.1 micrometers
  Real d0 = n / (6e-4); // gas density in units of 10^-27 g/cm^3
  Real T_0 = 2e6; // K
  Real omega = 2.5;
  Real A = 0.17e9 * YR_IN_S; // 0.17 Gyr in s

  return A * (a1/d0) * (pow(T_0/T, omega) + 1); // s
}

__device__ Real calc_dd_dt(Real d_dust, Real tau_sp) {
    return -d_dust / (tau_sp/3);
}

// function to initialize conserved variable array, similar to Grid3D::Constant in grid/initial_conditions.cpp 
void Conserved_Init(Real *host_conserved, Real rho, Real vx, Real vy, Real vz, Real P, Real rho_d, Real gamma, int n_cells, int nx, int ny, int nz, int n_ghost, int n_fields)
{
  int i, j, k, id;
  int istart, jstart, kstart, iend, jend, kend;

  istart = H.n_ghost;
  iend   = H.nx-H.n_ghost;
  if (H.ny > 1) {
    jstart = H.n_ghost;
    jend   = H.ny-H.n_ghost;
  }
  else {
    jstart = 0;
    jend   = H.ny;
  }
  if (H.nz > 1) {
    kstart = H.n_ghost;
    kend   = H.nz-H.n_ghost;
  }
  else {
    kstart = 0;
    kend   = H.nz;
  }

  // set initial values of conserved variables
  for(k=kstart-1; k<kend; k++) {
    for(j=jstart-1; j<jend; j++) {
      for(i=istart-1; i<iend; i++) {

        //get cell index
        id = i + j*nx + k*nx*ny;

        // Exclude the rightmost ghost cell on the "left" side
        if ((k >= kstart) and (j >= jstart) and (i >= istart))
        {
          // set constant initial states
          host_conserved[id] = rho;
          host_conserved[1*n_cells+id] = rho*vx;
          host_conserved[2*n_cells+id] = rho*vy;
          host_conserved[3*n_cells+id] = rho*vz;
          host_conserved[4*n_cells+id] = P/(gamma-1.0) + 0.5*rho*(vx*vx + vy*vy + vz*vz);
          #ifdef DE
          host_conserved[(n_fields-1)*n_cells+id] = P/(gamma-1.0);
          #endif  // DE
          #ifdef SCALAR
          host_conserved[5*n_cells+id] = rho_d;
          #endif // SCALAR
        }
      }
    }
  }
}


#endif // SCALAR
#endif // CUDA