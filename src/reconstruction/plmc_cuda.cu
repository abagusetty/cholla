#include "hip/hip_runtime.h"
/*! \file plmc_cuda.cu
 *  \brief Definitions of the piecewise linear reconstruction functions with
           limiting applied in the characteristic variables, as described
           in Stone et al., 2008. */

#include <math.h>

#include "../global/global.h"
#include "../global/global_cuda.h"
#include "../reconstruction/plmc_cuda.h"
#include "../utils/cuda_utilities.h"
#include "../utils/gpu.hpp"
#include "../utils/hydro_utilities.h"
#include "../utils/mhd_utilities.h"

#ifdef DE  // PRESSURE_DE
  #include "../utils/hydro_utilities.h"
#endif  // DE

/*! \fn __global__ void PLMC_cuda(Real *dev_conserved, Real *dev_bounds_L, Real
 *dev_bounds_R, int nx, int ny, int nz, Real dx, Real dt, Real
 gamma, int dir)
 *  \brief When passed a stencil of conserved variables, returns the left and
 right boundary values for the interface calculated using plm. */
__global__ void PLMC_cuda(Real *dev_conserved, Real *dev_bounds_L, Real *dev_bounds_R, int nx, int ny, int nz, Real dx,
                          Real dt, Real gamma, int dir, int n_fields)
{
  // get a thread ID
  int const thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int xid, yid, zid;
  cuda_utilities::compute3DIndices(thread_id, nx, ny, xid, yid, zid);

  // Thread guard to prevent overrun
  if (xid < 1 or xid >= nx - 2 or yid < 1 or yid >= ny - 2 or zid < 1 or zid >= nz - 2) {
    return;
  }

  // Compute the total number of cells
  int const n_cells = nx * ny * nz;

  // Set the field indices for the various directions
  int o1, o2, o3;
  switch (dir) {
    case 0:
      o1 = grid_enum::momentum_x;
      o2 = grid_enum::momentum_y;
      o3 = grid_enum::momentum_z;
      break;
    case 1:
      o1 = grid_enum::momentum_y;
      o2 = grid_enum::momentum_z;
      o3 = grid_enum::momentum_x;
      break;
    case 2:
      o1 = grid_enum::momentum_z;
      o2 = grid_enum::momentum_x;
      o3 = grid_enum::momentum_y;
      break;
  }

  // load the 3-cell stencil into registers
  // cell i
  plmc_utils::PlmcPrimitive const cell_i =
      plmc_utils::Load_Data(dev_conserved, xid, yid, zid, nx, ny, n_cells, o1, o2, o3, gamma);

  // cell i-1. The equality checks check the direction and subtract one from the direction
  plmc_utils::PlmcPrimitive const cell_imo = plmc_utils::Load_Data(
      dev_conserved, xid - int(dir == 0), yid - int(dir == 1), zid - int(dir == 2), nx, ny, n_cells, o1, o2, o3, gamma);

  // cell i+1. The equality checks check the direction and add one to the direction
  plmc_utils::PlmcPrimitive const cell_ipo = plmc_utils::Load_Data(
      dev_conserved, xid + int(dir == 0), yid + int(dir == 1), zid + int(dir == 2), nx, ny, n_cells, o1, o2, o3, gamma);

  // calculate the adiabatic sound speed in cell i
  Real const sound_speed         = hydro_utilities::Calc_Sound_Speed(cell_i.pressure, cell_i.density, gamma);
  Real const sound_speed_squared = sound_speed * sound_speed;

  // Compute the left, right, centered, and van Leer differences of the
  // primitive variables Note that here L and R refer to locations relative to
  // the cell center

  // left
  plmc_utils::PlmcPrimitive const del_L = plmc_utils::Compute_Slope(cell_i, cell_imo);

  // right
  plmc_utils::PlmcPrimitive const del_R = plmc_utils::Compute_Slope(cell_ipo, cell_i);

  // centered
  plmc_utils::PlmcPrimitive const del_C = plmc_utils::Compute_Slope(cell_ipo, cell_imo, 0.5);

  // Van Leer
  plmc_utils::PlmcPrimitive const del_G = plmc_utils::Van_Leer_Slope(del_L, del_R);

  // Project the left, right, centered and van Leer differences onto the
  // characteristic variables Stone Eqn 37 (del_a are differences in
  // characteristic variables, see Stone for notation) Use the eigenvectors
  // given in Stone 2008, Appendix A
  plmc_utils::PlmcCharacteristic del_a_L, del_a_R, del_a_C, del_a_G, del_a_m;
  del_a_L.a0 =
      -cell_i.density * del_L.velocity_x / (2 * sound_speed) + del_L.pressure / (2 * sound_speed * sound_speed);
  del_a_L.a1 = del_L.density - del_L.pressure / (sound_speed_squared);
  del_a_L.a2 = del_L.velocity_y;
  del_a_G.a3 = del_L.velocity_z;
  del_a_L.a4 = cell_i.density * del_L.velocity_x / (2 * sound_speed) + del_L.pressure / (2 * sound_speed_squared);

  del_a_R.a0 = -cell_i.density * del_R.velocity_x / (2 * sound_speed) + del_R.pressure / (2 * sound_speed_squared);
  del_a_R.a1 = del_R.density - del_R.pressure / (sound_speed_squared);
  del_a_R.a2 = del_R.velocity_y;
  del_a_R.a3 = del_R.velocity_z;
  del_a_R.a4 = cell_i.density * del_R.velocity_x / (2 * sound_speed) + del_R.pressure / (2 * sound_speed_squared);

  del_a_C.a0 = -cell_i.density * del_C.velocity_x / (2 * sound_speed) + del_C.pressure / (2 * sound_speed_squared);
  del_a_C.a1 = del_C.density - del_C.pressure / (sound_speed_squared);
  del_a_C.a2 = del_C.velocity_y;
  del_a_C.a3 = del_C.velocity_z;
  del_a_C.a4 = cell_i.density * del_C.velocity_x / (2 * sound_speed) + del_C.pressure / (2 * sound_speed_squared);

  del_a_G.a0 = -cell_i.density * del_G.velocity_x / (2 * sound_speed) + del_G.pressure / (2 * sound_speed_squared);
  del_a_G.a1 = del_G.density - del_G.pressure / (sound_speed_squared);
  del_a_G.a2 = del_G.velocity_y;
  del_a_G.a3 = del_G.velocity_z;
  del_a_G.a4 = cell_i.density * del_G.velocity_x / (2 * sound_speed) + del_G.pressure / (2 * sound_speed_squared);

  // Apply monotonicity constraints to the differences in the characteristic
  // variables

  del_a_m.a0 = del_a_m.a1 = del_a_m.a2 = del_a_m.a3 = del_a_m.a4 = 0.0;  // This should be in the declaration
  plmc_utils::PlmcPrimitive del_m_i;
  if (del_a_L.a0 * del_a_R.a0 > 0.0) {
    Real const lim_slope_a = fmin(fabs(del_a_L.a0), fabs(del_a_R.a0));
    Real const lim_slope_b = fmin(fabs(del_a_C.a0), fabs(del_a_G.a0));
    del_a_m.a0             = sgn_CUDA(del_a_C.a0) * fmin(2.0 * lim_slope_a, lim_slope_b);
  }
  if (del_a_L.a1 * del_a_R.a1 > 0.0) {
    Real const lim_slope_a = fmin(fabs(del_a_L.a1), fabs(del_a_R.a1));
    Real const lim_slope_b = fmin(fabs(del_a_C.a1), fabs(del_a_G.a1));
    del_a_m.a1             = sgn_CUDA(del_a_C.a1) * fmin(2.0 * lim_slope_a, lim_slope_b);
  }
  if (del_a_L.a2 * del_a_R.a2 > 0.0) {
    Real const lim_slope_a = fmin(fabs(del_a_L.a2), fabs(del_a_R.a2));
    Real const lim_slope_b = fmin(fabs(del_a_C.a2), fabs(del_a_G.a2));
    del_a_m.a2             = sgn_CUDA(del_a_C.a2) * fmin(2.0 * lim_slope_a, lim_slope_b);
  }
  if (del_a_G.a3 * del_a_R.a3 > 0.0) {
    Real const lim_slope_a = fmin(fabs(del_a_G.a3), fabs(del_a_R.a3));
    Real const lim_slope_b = fmin(fabs(del_a_C.a3), fabs(del_a_G.a3));
    del_a_m.a3             = sgn_CUDA(del_a_C.a3) * fmin(2.0 * lim_slope_a, lim_slope_b);
  }
  if (del_a_L.a4 * del_a_R.a4 > 0.0) {
    Real const lim_slope_a = fmin(fabs(del_a_L.a4), fabs(del_a_R.a4));
    Real const lim_slope_b = fmin(fabs(del_a_C.a4), fabs(del_a_G.a4));
    del_a_m.a4             = sgn_CUDA(del_a_C.a4) * fmin(2.0 * lim_slope_a, lim_slope_b);
  }
#ifdef DE
  del_m_i.gas_energy = 0.0;
  if (del_L.gas_energy * del_R.gas_energy > 0.0) {
    Real const lim_slope_a = fmin(fabs(del_L.gas_energy), fabs(del_R.gas_energy));
    Real const lim_slope_b = fmin(fabs(del_C.gas_energy), fabs(del_G.gas_energy));
    del_m_i.gas_energy     = sgn_CUDA(del_C.gas_energy) * fmin(2.0 * lim_slope_a, lim_slope_b);
  }
#endif  // DE
#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    del_m_i.scalar[i] = 0.0;
    if (del_L.scalar[i] * del_R.scalar[i] > 0.0) {
      Real const lim_slope_a = fmin(fabs(del_L.scalar[i]), fabs(del_R.scalar[i]));
      Real const lim_slope_b = fmin(fabs(del_C.scalar[i]), fabs(del_G.scalar[i]));
      del_m_i.scalar[i]      = sgn_CUDA(del_C.scalar[i]) * fmin(2.0 * lim_slope_a, lim_slope_b);
    }
  }
#endif  // SCALAR

  // Project the monotonized difference in the characteristic variables back
  // onto the primitive variables Stone Eqn 39
  del_m_i.density    = del_a_m.a0 + del_a_m.a1 + del_a_m.a4;
  del_m_i.velocity_x = -sound_speed * del_a_m.a0 / cell_i.density + sound_speed * del_a_m.a4 / cell_i.density;
  del_m_i.velocity_y = del_a_m.a2;
  del_m_i.velocity_z = del_a_m.a3;
  del_m_i.pressure   = sound_speed_squared * del_a_m.a0 + sound_speed_squared * del_a_m.a4;

  // Compute the left and right interface values using the monotonized
  // difference in the primitive variables

  plmc_utils::PlmcPrimitive interface_L_iph, interface_R_imh;

  interface_R_imh.density    = cell_i.density - 0.5 * del_m_i.density;
  interface_R_imh.velocity_x = cell_i.velocity_x - 0.5 * del_m_i.velocity_x;
  interface_R_imh.velocity_y = cell_i.velocity_y - 0.5 * del_m_i.velocity_y;
  interface_R_imh.velocity_z = cell_i.velocity_z - 0.5 * del_m_i.velocity_z;
  interface_R_imh.pressure   = cell_i.pressure - 0.5 * del_m_i.pressure;

  interface_L_iph.density    = cell_i.density + 0.5 * del_m_i.density;
  interface_L_iph.velocity_x = cell_i.velocity_x + 0.5 * del_m_i.velocity_x;
  interface_L_iph.velocity_y = cell_i.velocity_y + 0.5 * del_m_i.velocity_y;
  interface_L_iph.velocity_z = cell_i.velocity_z + 0.5 * del_m_i.velocity_z;
  interface_L_iph.pressure   = cell_i.pressure + 0.5 * del_m_i.pressure;

#ifdef DE
  interface_R_imh.gas_energy = cell_i.gas_energy - 0.5 * del_m_i.gas_energy;
  interface_L_iph.gas_energy = cell_i.gas_energy + 0.5 * del_m_i.gas_energy;
#endif  // DE
#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_R_imh.scalar[i] = cell_i.scalar[i] - 0.5 * del_m_i.scalar[i];
    interface_L_iph.scalar[i] = cell_i.scalar[i] + 0.5 * del_m_i.scalar[i];
  }
#endif  // SCALAR

  // try removing this on shock tubes
  Real C                  = interface_R_imh.density + interface_L_iph.density;
  interface_R_imh.density = fmax(fmin(cell_i.density, cell_imo.density), interface_R_imh.density);
  interface_R_imh.density = fmin(fmax(cell_i.density, cell_imo.density), interface_R_imh.density);
  interface_L_iph.density = C - interface_R_imh.density;
  interface_L_iph.density = fmax(fmin(cell_i.density, cell_ipo.density), interface_L_iph.density);
  interface_L_iph.density = fmin(fmax(cell_i.density, cell_ipo.density), interface_L_iph.density);
  interface_R_imh.density = C - interface_L_iph.density;

  C                          = interface_R_imh.velocity_x + interface_L_iph.velocity_x;
  interface_R_imh.velocity_x = fmax(fmin(cell_i.velocity_x, cell_imo.velocity_x), interface_R_imh.velocity_x);
  interface_R_imh.velocity_x = fmin(fmax(cell_i.velocity_x, cell_imo.velocity_x), interface_R_imh.velocity_x);
  interface_L_iph.velocity_x = C - interface_R_imh.velocity_x;
  interface_L_iph.velocity_x = fmax(fmin(cell_i.velocity_x, cell_ipo.velocity_x), interface_L_iph.velocity_x);
  interface_L_iph.velocity_x = fmin(fmax(cell_i.velocity_x, cell_ipo.velocity_x), interface_L_iph.velocity_x);
  interface_R_imh.velocity_x = C - interface_L_iph.velocity_x;

  C                          = interface_R_imh.velocity_y + interface_L_iph.velocity_y;
  interface_R_imh.velocity_y = fmax(fmin(cell_i.velocity_y, cell_imo.velocity_y), interface_R_imh.velocity_y);
  interface_R_imh.velocity_y = fmin(fmax(cell_i.velocity_y, cell_imo.velocity_y), interface_R_imh.velocity_y);
  interface_L_iph.velocity_y = C - interface_R_imh.velocity_y;
  interface_L_iph.velocity_y = fmax(fmin(cell_i.velocity_y, cell_ipo.velocity_y), interface_L_iph.velocity_y);
  interface_L_iph.velocity_y = fmin(fmax(cell_i.velocity_y, cell_ipo.velocity_y), interface_L_iph.velocity_y);
  interface_R_imh.velocity_y = C - interface_L_iph.velocity_y;

  C                          = interface_R_imh.velocity_z + interface_L_iph.velocity_z;
  interface_R_imh.velocity_z = fmax(fmin(cell_i.velocity_z, cell_imo.velocity_z), interface_R_imh.velocity_z);
  interface_R_imh.velocity_z = fmin(fmax(cell_i.velocity_z, cell_imo.velocity_z), interface_R_imh.velocity_z);
  interface_L_iph.velocity_z = C - interface_R_imh.velocity_z;
  interface_L_iph.velocity_z = fmax(fmin(cell_i.velocity_z, cell_ipo.velocity_z), interface_L_iph.velocity_z);
  interface_L_iph.velocity_z = fmin(fmax(cell_i.velocity_z, cell_ipo.velocity_z), interface_L_iph.velocity_z);
  interface_R_imh.velocity_z = C - interface_L_iph.velocity_z;

  C                        = interface_R_imh.pressure + interface_L_iph.pressure;
  interface_R_imh.pressure = fmax(fmin(cell_i.pressure, cell_imo.pressure), interface_R_imh.pressure);
  interface_R_imh.pressure = fmin(fmax(cell_i.pressure, cell_imo.pressure), interface_R_imh.pressure);
  interface_L_iph.pressure = C - interface_R_imh.pressure;
  interface_L_iph.pressure = fmax(fmin(cell_i.pressure, cell_ipo.pressure), interface_L_iph.pressure);
  interface_L_iph.pressure = fmin(fmax(cell_i.pressure, cell_ipo.pressure), interface_L_iph.pressure);
  interface_R_imh.pressure = C - interface_L_iph.pressure;

  del_m_i.density    = interface_L_iph.density - interface_R_imh.density;
  del_m_i.velocity_x = interface_L_iph.velocity_x - interface_R_imh.velocity_x;
  del_m_i.velocity_y = interface_L_iph.velocity_y - interface_R_imh.velocity_y;
  del_m_i.velocity_z = interface_L_iph.velocity_z - interface_R_imh.velocity_z;
  del_m_i.pressure   = interface_L_iph.pressure - interface_R_imh.pressure;

#ifdef DE
  C                          = interface_R_imh.gas_energy + interface_L_iph.gas_energy;
  interface_R_imh.gas_energy = fmax(fmin(cell_i.gas_energy, cell_imo.gas_energy), interface_R_imh.gas_energy);
  interface_R_imh.gas_energy = fmin(fmax(cell_i.gas_energy, cell_imo.gas_energy), interface_R_imh.gas_energy);
  interface_L_iph.gas_energy = C - interface_R_imh.gas_energy;
  interface_L_iph.gas_energy = fmax(fmin(cell_i.gas_energy, cell_ipo.gas_energy), interface_L_iph.gas_energy);
  interface_L_iph.gas_energy = fmin(fmax(cell_i.gas_energy, cell_ipo.gas_energy), interface_L_iph.gas_energy);
  interface_R_imh.gas_energy = C - interface_L_iph.gas_energy;
  del_m_i.gas_energy         = interface_L_iph.gas_energy - interface_R_imh.gas_energy;
#endif  // DE

#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    C                         = interface_R_imh.scalar[i] + interface_L_iph.scalar[i];
    interface_R_imh.scalar[i] = fmax(fmin(cell_i.scalar[i], cell_imo.scalar[i]), interface_R_imh.scalar[i]);
    interface_R_imh.scalar[i] = fmin(fmax(cell_i.scalar[i], cell_imo.scalar[i]), interface_R_imh.scalar[i]);
    interface_L_iph.scalar[i] = C - interface_R_imh.scalar[i];
    interface_L_iph.scalar[i] = fmax(fmin(cell_i.scalar[i], cell_ipo.scalar[i]), interface_L_iph.scalar[i]);
    interface_L_iph.scalar[i] = fmin(fmax(cell_i.scalar[i], cell_ipo.scalar[i]), interface_L_iph.scalar[i]);
    interface_R_imh.scalar[i] = C - interface_L_iph.scalar[i];
    del_m_i.scalar[i]         = interface_L_iph.scalar[i] - interface_R_imh.scalar[i];
  }
#endif  // SCALAR

#ifndef VL

  Real const dtodx = dt / dx;

  // Compute the eigenvalues of the linearized equations in the
  // primitive variables using the cell-centered primitive variables
  Real const lambda_m = cell_i.velocity_x - sound_speed;
  Real const lambda_0 = cell_i.velocity_x;
  Real const lambda_p = cell_i.velocity_x + sound_speed;

  // Integrate linear interpolation function over domain of dependence
  // defined by max(min) eigenvalue
  Real qx                    = -0.5 * fmin(lambda_m, 0.0) * dtodx;
  interface_R_imh.density    = interface_R_imh.density + qx * del_m_i.density;
  interface_R_imh.velocity_x = interface_R_imh.velocity_x + qx * del_m_i.velocity_x;
  interface_R_imh.velocity_y = interface_R_imh.velocity_y + qx * del_m_i.velocity_y;
  interface_R_imh.velocity_z = interface_R_imh.velocity_z + qx * del_m_i.velocity_z;
  interface_R_imh.pressure   = interface_R_imh.pressure + qx * del_m_i.pressure;

  qx                         = 0.5 * fmax(lambda_p, 0.0) * dtodx;
  interface_L_iph.density    = interface_L_iph.density - qx * del_m_i.density;
  interface_L_iph.velocity_x = interface_L_iph.velocity_x - qx * del_m_i.velocity_x;
  interface_L_iph.velocity_y = interface_L_iph.velocity_y - qx * del_m_i.velocity_y;
  interface_L_iph.velocity_z = interface_L_iph.velocity_z - qx * del_m_i.velocity_z;
  interface_L_iph.pressure   = interface_L_iph.pressure - qx * del_m_i.pressure;

  #ifdef DE
  interface_R_imh.gas_energy = interface_R_imh.gas_energy + qx * del_m_i.gas_energy;
  interface_L_iph.gas_energy = interface_L_iph.gas_energy - qx * del_m_i.gas_energy;
  #endif  // DE

  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_R_imh.scalar[i] = interface_R_imh.scalar[i] + qx * del_m_i.scalar[i];
    interface_L_iph.scalar[i] = interface_L_iph.scalar[i] - qx * del_m_i.scalar[i];
  }
  #endif  // SCALAR

  // Perform the characteristic tracing
  // Stone Eqns 42 & 43

  // left-hand interface value, i+1/2
  Real sum_0 = 0.0, sum_1 = 0.0, sum_2 = 0.0, sum_3 = 0.0, sum_4 = 0.0;
  #ifdef DE
  Real sum_ge = 0;
  #endif  // DE
  #ifdef SCALAR
  Real sum_scalar[NSCALARS];
  for (int i = 0; i < NSCALARS; i++) {
    sum_scalar[i] = 0.0;
  }
  #endif  // SCALAR
  if (lambda_m >= 0) {
    Real lamdiff = lambda_p - lambda_m;

    sum_0 += lamdiff *
             (-cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 - del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (-cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }
  if (lambda_0 >= 0) {
    Real lamdiff = lambda_p - lambda_0;

    sum_0 += lamdiff * (del_m_i.density - del_m_i.pressure / (sound_speed_squared));
    sum_2 += lamdiff * del_m_i.velocity_y;
    sum_3 += lamdiff * del_m_i.velocity_z;
  #ifdef DE
    sum_ge += lamdiff * del_m_i.gas_energy;
  #endif  // DE
  #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      sum_scalar[i] += lamdiff * del_m_i.scalar[i];
    }
  #endif  // SCALAR
  }
  if (lambda_p >= 0) {
    Real lamdiff = lambda_p - lambda_p;

    sum_0 += lamdiff *
             (cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 + del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }

  // add the corrections to the initial guesses for the interface values
  interface_L_iph.density += 0.5 * dtodx * sum_0;
  interface_L_iph.velocity_x += 0.5 * dtodx * sum_1;
  interface_L_iph.velocity_y += 0.5 * dtodx * sum_2;
  interface_L_iph.velocity_z += 0.5 * dtodx * sum_3;
  interface_L_iph.pressure += 0.5 * dtodx * sum_4;
  #ifdef DE
  interface_L_iph.gas_energy += 0.5 * dtodx * sum_ge;
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_L_iph.scalar[i] += 0.5 * dtodx * sum_scalar[i];
  }
  #endif  // SCALAR

  // right-hand interface value, i-1/2
  sum_0 = sum_1 = sum_2 = sum_3 = sum_4 = 0;
  #ifdef DE
  sum_ge = 0;
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    sum_scalar[i] = 0;
  }
  #endif  // SCALAR
  if (lambda_m <= 0) {
    Real lamdiff = lambda_m - lambda_m;

    sum_0 += lamdiff *
             (-cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 - del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (-cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }
  if (lambda_0 <= 0) {
    Real lamdiff = lambda_m - lambda_0;

    sum_0 += lamdiff * (del_m_i.density - del_m_i.pressure / (sound_speed_squared));
    sum_2 += lamdiff * del_m_i.velocity_y;
    sum_3 += lamdiff * del_m_i.velocity_z;
  #ifdef DE
    sum_ge += lamdiff * del_m_i.gas_energy;
  #endif  // DE
  #ifdef SCALAR
    for (int i = 0; i < NSCALARS; i++) {
      sum_scalar[i] += lamdiff * del_m_i.scalar[i];
    }
  #endif  // SCALAR
  }
  if (lambda_p <= 0) {
    Real lamdiff = lambda_m - lambda_p;

    sum_0 += lamdiff *
             (cell_i.density * del_m_i.velocity_x / (2 * sound_speed) + del_m_i.pressure / (2 * sound_speed_squared));
    sum_1 += lamdiff * (del_m_i.velocity_x / 2.0 + del_m_i.pressure / (2 * sound_speed * cell_i.density));
    sum_4 += lamdiff * (cell_i.density * del_m_i.velocity_x * sound_speed / 2.0 + del_m_i.pressure / 2.0);
  }

  // add the corrections
  interface_R_imh.density += 0.5 * dtodx * sum_0;
  interface_R_imh.velocity_x += 0.5 * dtodx * sum_1;
  interface_R_imh.velocity_y += 0.5 * dtodx * sum_2;
  interface_R_imh.velocity_z += 0.5 * dtodx * sum_3;
  interface_R_imh.pressure += 0.5 * dtodx * sum_4;
  #ifdef DE
  interface_R_imh.gas_energy += 0.5 * dtodx * sum_ge;
  #endif  // DE
  #ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    interface_R_imh.scalar[i] += 0.5 * dtodx * sum_scalar[i];
  }
  #endif  // SCALAR
#endif    // CTU

  // apply minimum constraints
  interface_R_imh.density  = fmax(interface_R_imh.density, (Real)TINY_NUMBER);
  interface_L_iph.density  = fmax(interface_L_iph.density, (Real)TINY_NUMBER);
  interface_R_imh.pressure = fmax(interface_R_imh.pressure, (Real)TINY_NUMBER);
  interface_L_iph.pressure = fmax(interface_L_iph.pressure, (Real)TINY_NUMBER);

  // Convert the left and right states in the primitive to the conserved
  // variables send final values back from kernel bounds_R refers to the right
  // side of the i-1/2 interface
  int id;
  switch (dir) {
    case 0:
      id = xid - 1 + yid * nx + zid * nx * ny;
      break;
    case 1:
      id = xid + (yid - 1) * nx + zid * nx * ny;
      break;
    case 2:
      id = xid + yid * nx + (zid - 1) * nx * ny;
      break;
  }

  dev_bounds_R[id]                = interface_R_imh.density;
  dev_bounds_R[o1 * n_cells + id] = interface_R_imh.density * interface_R_imh.velocity_x;
  dev_bounds_R[o2 * n_cells + id] = interface_R_imh.density * interface_R_imh.velocity_y;
  dev_bounds_R[o3 * n_cells + id] = interface_R_imh.density * interface_R_imh.velocity_z;
  dev_bounds_R[4 * n_cells + id] =
      (interface_R_imh.pressure / (gamma - 1.0)) + 0.5 * interface_R_imh.density *
                                                       (interface_R_imh.velocity_x * interface_R_imh.velocity_x +
                                                        interface_R_imh.velocity_y * interface_R_imh.velocity_y +
                                                        interface_R_imh.velocity_z * interface_R_imh.velocity_z);
#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    dev_bounds_R[(5 + i) * n_cells + id] = interface_R_imh.density * interface_R_imh.scalar[i];
  }
#endif  // SCALAR
#ifdef DE
  dev_bounds_R[(n_fields - 1) * n_cells + id] = interface_R_imh.density * interface_R_imh.gas_energy;
#endif  // DE
  // bounds_L refers to the left side of the i+1/2 interface
  id                              = xid + yid * nx + zid * nx * ny;
  dev_bounds_L[id]                = interface_L_iph.density;
  dev_bounds_L[o1 * n_cells + id] = interface_L_iph.density * interface_L_iph.velocity_x;
  dev_bounds_L[o2 * n_cells + id] = interface_L_iph.density * interface_L_iph.velocity_y;
  dev_bounds_L[o3 * n_cells + id] = interface_L_iph.density * interface_L_iph.velocity_z;
  dev_bounds_L[4 * n_cells + id] =
      (interface_L_iph.pressure / (gamma - 1.0)) + 0.5 * interface_L_iph.density *
                                                       (interface_L_iph.velocity_x * interface_L_iph.velocity_x +
                                                        interface_L_iph.velocity_y * interface_L_iph.velocity_y +
                                                        interface_L_iph.velocity_z * interface_L_iph.velocity_z);
#ifdef SCALAR
  for (int i = 0; i < NSCALARS; i++) {
    dev_bounds_L[(5 + i) * n_cells + id] = interface_L_iph.density * interface_L_iph.scalar[i];
  }
#endif  // SCALAR
#ifdef DE
  dev_bounds_L[(n_fields - 1) * n_cells + id] = interface_L_iph.density * interface_L_iph.gas_energy;
#endif  // DE
}

namespace plmc_utils
{
// =====================================================================================================================
PlmcPrimitive __device__ __host__ Load_Data(Real const *dev_conserved, size_t const &xid, size_t const &yid,
                                            size_t const &zid, size_t const &nx, size_t const &ny,
                                            size_t const &n_cells, size_t const &o1, size_t const &o2, size_t const &o3,
                                            Real const &gamma)
{
  // Compute index
  size_t const id = cuda_utilities::compute1DIndex(xid, yid, zid, nx, ny);

  // Declare the variable we will return
  PlmcPrimitive loaded_data;

  // Load hydro variables except pressure
  loaded_data.density    = dev_conserved[grid_enum::density * n_cells + id];
  loaded_data.velocity_x = dev_conserved[o1 * n_cells + id] / loaded_data.density;
  loaded_data.velocity_y = dev_conserved[o2 * n_cells + id] / loaded_data.density;
  loaded_data.velocity_z = dev_conserved[o3 * n_cells + id] / loaded_data.density;

  // Load MHD variables. Note that I only need the centered values for the transverse fields except for the initial
  // computation of the primitive variables
#ifdef MHD
  auto magnetic_centered = mhd::utils::cellCenteredMagneticFields(dev_conserved, id, xid, yid, zid, n_cells, nx, ny);
  switch (o1) {
    case grid_enum::momentum_x:
      loaded_data.magnetic_y = magnetic_centered.y;
      loaded_data.magnetic_z = magnetic_centered.z;
      break;
    case grid_enum::momentum_y:
      loaded_data.magnetic_y = magnetic_centered.z;
      loaded_data.magnetic_z = magnetic_centered.x;
      break;
    case grid_enum::momentum_z:
      loaded_data.magnetic_y = magnetic_centered.x;
      loaded_data.magnetic_z = magnetic_centered.y;
      break;
  }
#endif  // MHD

// Load pressure accounting for duel energy if enabled
#ifdef DE  // DE
  Real const E          = dev_conserved[grid_enum::Energy * n_cells + id];
  Real const gas_energy = dev_conserved[grid_enum::GasEnergy * n_cells + id];

  Real E_non_thermal = hydro_utilities::Calc_Kinetic_Energy_From_Velocity(
      loaded_data.density, loaded_data.velocity_x, loaded_data.velocity_y, loaded_data.velocity_z);

  #ifdef MHD
  E_non_thermal += mhd::utils::computeMagneticEnergy(magnetic_centered.x, magnetic_centered.y, magnetic_centered.z);
  #endif  // MHD

  loaded_data.pressure   = hydro_utilities::Get_Pressure_From_DE(E, E - E_non_thermal, gas_energy, gamma);
  loaded_data.gas_energy = gas_energy / loaded_data.density;
#else  // not DE
  #ifdef MHD
  loaded_data.pressure =
      hydro_utilities::Calc_Pressure_Primitive(dev_conserved[grid_enum::Energy * n_cells + id], loaded_data.density,
                                               loaded_data.velocity_x, loaded_data.velocity_y, loaded_data.velocity_z,
                                               gamma, magnetic_centered.x, magnetic_centered.y, magnetic_centered.z);
  #else   // not MHD
  loaded_data.pressure = hydro_utilities::Calc_Pressure_Primitive(
      dev_conserved[grid_enum::Energy * n_cells + id], loaded_data.density, loaded_data.velocity_x,
      loaded_data.velocity_y, loaded_data.velocity_z, gamma);
  #endif  // MHD
#endif    // DE

#ifdef SCALAR
  for (size_t i = 0; i < grid_enum::nscalars; i++) {
    loaded_data.scalar[i] = dev_conserved[(grid_enum::scalar + i) * n_cells + id] / loaded_data.density;
  }
#endif  // SCALAR

  return loaded_data;
}
// =====================================================================================================================

// =====================================================================================================================
PlmcPrimitive __device__ __host__ Compute_Slope(PlmcPrimitive const &left, PlmcPrimitive const &right, Real const &coef)
{
  PlmcPrimitive slopes;

  slopes.density    = coef * (left.density - right.density);
  slopes.velocity_x = coef * (left.velocity_x - right.velocity_x);
  slopes.velocity_y = coef * (left.velocity_y - right.velocity_y);
  slopes.velocity_z = coef * (left.velocity_z - right.velocity_z);
  slopes.pressure   = coef * (left.pressure - right.pressure);

#ifdef MHD
  slopes.magnetic_y = coef * (left.magnetic_y - right.magnetic_y);
  slopes.magnetic_z = coef * (left.magnetic_z - right.magnetic_z);
#endif  // MHD

#ifdef DE
  slopes.gas_energy = coef * (left.gas_energy - right.gas_energy);
#endif  // DE

#ifdef SCALAR
  for (size_t i = 0; i < grid_enum::nscalars; i++) {
    slopes.scalar[i] = coef * (left.scalar[i] - right.scalar[i]);
  }
#endif  // SCALAR

  return slopes;
}
// =====================================================================================================================

// =====================================================================================================================
PlmcPrimitive __device__ __host__ Van_Leer_Slope(PlmcPrimitive const &left_slope, PlmcPrimitive const &right_slope)
{
  PlmcPrimitive vl_slopes;

  auto Calc_Vl_Slope = [](Real const &left, Real const &right) -> Real {
    if (left * right > 0.0) {
      return 2.0 * left * right / (left + right);
    } else {
      return 0.0;
    }
  };

  vl_slopes.density    = Calc_Vl_Slope(left_slope.density, right_slope.density);
  vl_slopes.velocity_x = Calc_Vl_Slope(left_slope.velocity_x, right_slope.velocity_x);
  vl_slopes.velocity_y = Calc_Vl_Slope(left_slope.velocity_y, right_slope.velocity_y);
  vl_slopes.velocity_z = Calc_Vl_Slope(left_slope.velocity_z, right_slope.velocity_z);
  vl_slopes.pressure   = Calc_Vl_Slope(left_slope.pressure, right_slope.pressure);

#ifdef MHD
  vl_slopes.magnetic_y = Calc_Vl_Slope(left_slope.magnetic_y, right_slope.magnetic_y);
  vl_slopes.magnetic_z = Calc_Vl_Slope(left_slope.magnetic_z, right_slope.magnetic_z);
#endif  // MHD

#ifdef DE
  vl_slopes.gas_energy = Calc_Vl_Slope(left_slope.gas_energy, right_slope.gas_energy);
#endif  // DE

#ifdef SCALAR
  for (size_t i = 0; i < grid_enum::nscalars; i++) {
    vl_slopes.scalar[i] = Calc_Vl_Slope(left_slope.scalar[i], right_slope.scalar[i]);
  }
#endif  // SCALAR

  return vl_slopes;
}
// =====================================================================================================================
}  // namespace plmc_utils
